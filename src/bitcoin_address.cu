#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <secp256k1.h>
#include "sha256_cuda.cuh"
#include "ripemd160_cuda.cuh"
#include "GPU/GPUConstants.h"
#include "CPU/Int.h"
#include "CPU/Point.h"
#include "CPU/SECP256k1.h"
#include "GPU/GPUSecp.h"

// Variáveis globais para tabelas G na GPU
uint8_t *d_gTableX = nullptr;
uint8_t *d_gTableY = nullptr;

// Função para carregar as tabelas G
void loadGTable(uint8_t *gTableX, uint8_t *gTableY) {
    // Alocar memória temporária na CPU
    uint8_t *hostTableX = new uint8_t[NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE * SIZE_GTABLE_POINT];
    uint8_t *hostTableY = new uint8_t[NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE * SIZE_GTABLE_POINT];

    // Gerar tabelas na CPU
    Secp256K1 *secp = new Secp256K1();
    secp->Init();

    for (int i = 0; i < NUM_GTABLE_CHUNK; i++) {
        for (int j = 0; j < NUM_GTABLE_VALUE - 1; j++) {
            int element = (i * NUM_GTABLE_VALUE) + j;
            Point p = secp->GTable[element];
            for (int b = 0; b < 32; b++) {
                hostTableX[(element * SIZE_GTABLE_POINT) + b] = p.x.GetByte64(b);
                hostTableY[(element * SIZE_GTABLE_POINT) + b] = p.y.GetByte64(b);
            }
        }
    }

    delete secp;

    hipError_t err;
    err = hipMemcpy(gTableX, hostTableX, 
                     NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE * SIZE_GTABLE_POINT, 
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Erro ao copiar gTableX para GPU: %s\n", hipGetErrorString(err));
    }

    err = hipMemcpy(gTableY, hostTableY, 
                     NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE * SIZE_GTABLE_POINT, 
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Erro ao copiar gTableY para GPU: %s\n", hipGetErrorString(err));
    }

    delete[] hostTableX;
    delete[] hostTableY;
}

void freeGPUTables() {
    if (d_gTableX) hipFree(d_gTableX);
    if (d_gTableY) hipFree(d_gTableY);
    d_gTableX = nullptr;
    d_gTableY = nullptr;
}

bool initGPUTables() {
    hipError_t err;
    size_t tableSize = NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE * SIZE_GTABLE_POINT;
    printf("Alocando %zu bytes para cada tabela...\n", tableSize);

    err = hipMalloc(&d_gTableX, tableSize);
    if (err != hipSuccess) {
        printf("Erro ao alocar memória para gTableX: %s\n", hipGetErrorString(err));
        return false;
    }

    err = hipMalloc(&d_gTableY, tableSize);
    if (err != hipSuccess) {
        printf("Erro ao alocar memória para gTableY: %s\n", hipGetErrorString(err));
        hipFree(d_gTableX);
        return false;
    }

    printf("Carregando tabelas...\n");
    loadGTable(d_gTableX, d_gTableY);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Erro após carregar tabelas: %s\n", hipGetErrorString(err));
        freeGPUTables();
        return false;
    }

    printf("Tabelas carregadas com sucesso!\n");
    return true;
}

// Função device para incrementar a chave privada
__device__ void increment_private_key_gpu(unsigned char *private_key, uint64_t increment) {
    uint64_t carry = increment;
    for (int i = 31; i >= 0 && carry > 0; i--) {
        uint64_t sum = (uint64_t)private_key[i] + carry;
        private_key[i] = sum & 0xFF;
        carry = sum >> 8;
    }
}

// Kernel otimizado para processar múltiplas chaves em paralelo
__global__ void bitcoin_address_kernel(unsigned char* private_key, unsigned char* bitcoin_address, 
                                     const unsigned char* target_address, int* match_found,
                                     uint8_t* gTableX, uint8_t* gTableY,
                                     int keys_per_thread) {
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Memória compartilhada para tabelas G frequentemente acessadas
    __shared__ uint8_t shared_gTableX[1024];
    __shared__ uint8_t shared_gTableY[1024];
    
    // Carregar dados frequentemente acessados na memória compartilhada
    if (threadIdx.x < 1024) {
        shared_gTableX[threadIdx.x] = gTableX[threadIdx.x];
        shared_gTableY[threadIdx.x] = gTableY[threadIdx.x];
    }
    __syncthreads();

    for(int i = 0; i < keys_per_thread && !(*match_found); i++) {
        // Buffer local para a chave privada
        unsigned char local_private_key[32];
        memcpy(local_private_key, private_key, 32);
        
        // Incrementa a chave baseado no thread ID e iteração
        increment_private_key_gpu(local_private_key, tid + (i * gridDim.x * blockDim.x));

        // Buffers para as hashes
        unsigned char sha256_hash[SHA256_DIGEST_SIZE];
        unsigned char ripemd160_hash[RIPEMD160_DIGEST_SIZE];
        unsigned char public_key[33];  // Compressed public key format

        // Converter a chave privada para o formato correto
        uint16_t privKeyChunks[NUM_GTABLE_CHUNK] = {0};
        
        // Converter similar ao kernel host
        uint16_t* privKeyShorts = (uint16_t*)local_private_key;
        for (int j = 0; j < 16; j++) {
            uint16_t value = privKeyShorts[j];
            value = ((value & 0xFF00) >> 8) | ((value & 0x00FF) << 8);
            privKeyChunks[15 - j] = value;
        }

        // Gerar public key usando as tabelas G
        uint64_t pubX[4], pubY[4];
        _PointMultiSecp256k1(pubX, pubY, privKeyChunks, gTableX, gTableY);

        // Converter para formato comprimido (33 bytes)
        public_key[0] = 0x02 | (pubY[0] & 1);
        for (int j = 0; j < 32; j++) {
            public_key[j+1] = ((unsigned char*)pubX)[31-j];
        }

        // Calcular hashes
        sha256_gpu(public_key, 33, sha256_hash);
        ripemd160_gpu(sha256_hash, SHA256_DIGEST_SIZE, ripemd160_hash);

        // Verificar match
        bool match = true;
        for (int j = 0; j < RIPEMD160_DIGEST_SIZE; j++) {
            if (ripemd160_hash[j] != target_address[j]) {
                match = false;
                break;
            }
        }

        if (match) {
            atomicExch(match_found, 1);
            memcpy(bitcoin_address, ripemd160_hash, RIPEMD160_DIGEST_SIZE);
            memcpy(private_key, local_private_key, 32);
        }
    }
}

// Função auxiliar para converter hex string para bytes
bool hex_to_bytes(const char* hex_str, unsigned char* bytes, size_t length) {
    if (strlen(hex_str) != length * 2) return false;
    
    for (size_t i = 0; i < length; i++) {
        char hex_byte[3] = {hex_str[i*2], hex_str[i*2+1], 0};
        char* end_ptr;
        bytes[i] = (unsigned char)strtol(hex_byte, &end_ptr, 16);
        if (*end_ptr != 0) return false;
    }
    return true;
}

// Adicionar a função formatSpeed
const char* formatSpeed(double speed) {
    static char buffer[16];
    if (speed >= 1e9) {
        snprintf(buffer, sizeof(buffer), "%.2f Gkeys/s", speed / 1e9);
    } else if (speed >= 1e6) {
        snprintf(buffer, sizeof(buffer), "%.2f Mkeys/s", speed / 1e6);
    } else if (speed >= 1e3) {
        snprintf(buffer, sizeof(buffer), "%.2f Kkeys/s", speed / 1e3);
    } else {
        snprintf(buffer, sizeof(buffer), "%.2f keys/s", speed);
    }
    return buffer;
}

int main(int argc, char **argv) {
    int blockSize = 256;  // default
    int numBlocks = 4096; // aumentado
    int numStreams = 8;   // default
    int keysPerThread = 1;  // default
    const int BATCH_SIZE = 16; // novo parâmetro para batch processing
    
    // Chave privada default
    unsigned char private_key[32] = {
        0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
        0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
        0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x02,
        0x83, 0x2E, 0xD7, 0x0F, 0x2B, 0x5C, 0x35, 0xEE
    };

    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-block") == 0 && i + 1 < argc) {
            blockSize = atoi(argv[i + 1]);
            i++;
        }
        else if (strcmp(argv[i], "-grid") == 0 && i + 1 < argc) {
            numBlocks = atoi(argv[i + 1]);
            i++;
        }
        else if (strcmp(argv[i], "-streams") == 0 && i + 1 < argc) {
            numStreams = atoi(argv[i + 1]);
            i++;
        }
        else if (strcmp(argv[i], "-keys") == 0 && i + 1 < argc) {
            keysPerThread = atoi(argv[i + 1]);
            i++;
        }
        else if (strcmp(argv[i], "-private") == 0 && i + 1 < argc) {
            if (!hex_to_bytes(argv[i + 1], private_key, 32)) {
                printf("Erro: chave privada inválida. Use 64 caracteres hexadecimais\n");
                return 1;
            }
            i++;
        }
    }

    // Validar parâmetros
    if (keysPerThread <= 0 || keysPerThread > 16) {
        printf("Erro: keys per thread deve estar entre 1 e 16\n");
        return 1;
    }

    // Validar parâmetros
    if (numStreams <= 0 || numStreams > 32) {
        printf("Erro: número de streams deve estar entre 1 e 32\n");
        return 1;
    }

    // Validar parâmetros
    if (blockSize <= 0 || blockSize > 1024) {
        printf("Erro: block size deve estar entre 1 e 1024\n");
        return 1;
    }
    if (numBlocks <= 0) {
        printf("Erro: grid size deve ser maior que 0\n");
        return 1;
    }

    printf("Configuração:\n");
    printf("Block Size: %d\n", blockSize);
    printf("Grid Size: %d\n", numBlocks);
    printf("Num Streams: %d\n", numStreams);
    printf("Keys per Thread: %d\n", keysPerThread);
    printf("Total Threads: %d\n", blockSize * numBlocks);
    printf("Private Key Inicial: ");
    for (int i = 0; i < 32; i++) {
        printf("%02x", private_key[i]);
    }
    printf("\n");

    printf("Inicializando tabelas G na GPU...\n");
    if (!initGPUTables()) {
        printf("Falha ao inicializar tabelas G. Abortando.\n");
        return 1;
    }

    unsigned char bitcoin_address[RIPEMD160_DIGEST_SIZE];
    unsigned char target_bitcoin_address[RIPEMD160_DIGEST_SIZE] = {
        0x20, 0xd4, 0x5a, 0x6a, 0x76, 0x25, 0x35, 0x70, 
        0x0c, 0xe9, 0xe0, 0xb2, 0x16, 0xe3, 0x19, 0x94, 
        0x33, 0x5d, 0xb8, 0xa5  
    };

    // Alocar memória pinned na CPU para transferências mais rápidas
    unsigned char *host_private_key;
    unsigned char *host_bitcoin_address;
    hipHostAlloc(&host_private_key, 32, hipHostMallocDefault);
    hipHostAlloc(&host_bitcoin_address, RIPEMD160_DIGEST_SIZE, hipHostMallocDefault);
    memcpy(host_private_key, private_key, 32);

    // Alocar memória para o endereço target (compartilhado entre streams)
    unsigned char *d_target_address;
    hipMalloc(&d_target_address, RIPEMD160_DIGEST_SIZE);

    const int NUM_STREAMS = numStreams;
    const int KEYS_PER_THREAD = keysPerThread;
    CUDAStream streams[NUM_STREAMS];

    // Inicializar streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i].stream);
        hipMalloc(&streams[i].d_private_key, 32);
        hipMalloc(&streams[i].d_bitcoin_address, RIPEMD160_DIGEST_SIZE);
        hipMalloc(&streams[i].d_match_found, sizeof(int));
        streams[i].in_use = false;
    }

    hipMemcpyAsync(d_target_address, target_bitcoin_address, 
                    RIPEMD160_DIGEST_SIZE, hipMemcpyHostToDevice, streams[0].stream);

    const uint64_t TOTAL_THREADS = blockSize * numBlocks;
    const uint64_t TOTAL_THREADS_PER_BATCH = TOTAL_THREADS * BATCH_SIZE;
    int current_stream = 0;
    
    // Variáveis para estatísticas
    int match_found_host = 0;
    clock_t start_time = clock();
    uint64_t addresses_processed = 0;
    uint64_t total_addresses_processed = 0;
    int display_interval = 1;
    clock_t search_start_time = clock();

    while (!match_found_host) {
        // Verificar streams completados
        for (int i = 0; i < NUM_STREAMS; i++) {
            if (streams[i].in_use) {
                hipError_t err = hipStreamQuery(streams[i].stream);
                if (err == hipSuccess) {
                    int stream_match = 0;
                    hipMemcpyAsync(&stream_match, streams[i].d_match_found,
                                   sizeof(int), hipMemcpyDeviceToHost, streams[i].stream);
                    
                    if (stream_match) {
                        match_found_host = 1;
                        current_stream = i;
                        break;
                    }
                    streams[i].in_use = false;
                }
            }
        }

        if (streams[current_stream].in_use) {
            continue;
        }

        hipMemsetAsync(streams[current_stream].d_match_found, 0, sizeof(int), 
                       streams[current_stream].stream);

        hipMemcpyAsync(streams[current_stream].d_private_key, host_private_key,
                       32, hipMemcpyHostToDevice, streams[current_stream].stream);

        bitcoin_address_kernel<<<numBlocks, blockSize, 0, streams[current_stream].stream>>>(
            streams[current_stream].d_private_key,
            streams[current_stream].d_bitcoin_address,
            d_target_address,
            streams[current_stream].d_match_found,
            d_gTableX,
            d_gTableY,
            keysPerThread
        );

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Erro no kernel: %s\n", hipGetErrorString(err));
        }

        streams[current_stream].in_use = true;

        // Incrementar a chave para o próximo batch
        uint64_t carry = TOTAL_THREADS * KEYS_PER_THREAD;
        for (int i = 31; i >= 0 && carry > 0; i--) {
            uint64_t sum = (uint64_t)host_private_key[i] + carry;
            host_private_key[i] = sum & 0xFF;
            carry = sum >> 8;
        }

        addresses_processed += TOTAL_THREADS * KEYS_PER_THREAD;
        total_addresses_processed += TOTAL_THREADS * KEYS_PER_THREAD;

        // Estatísticas de performance
        clock_t current_time = clock();
        double elapsed_time = (double)(current_time - start_time) / CLOCKS_PER_SEC;
        if (elapsed_time >= display_interval) {
            double keys_per_second = addresses_processed / elapsed_time;
            printf("\rVelocidade: %s | Total processado: %lu", 
                   formatSpeed(keys_per_second), 
                   total_addresses_processed);
            fflush(stdout);
            start_time = clock();
            addresses_processed = 0;
        }

        current_stream = (current_stream + 1) % NUM_STREAMS;
    }

    clock_t search_end_time = clock();
    double total_search_time = (double)(search_end_time - search_start_time) / CLOCKS_PER_SEC;
    double avg_speed = total_addresses_processed / total_search_time;

    if (match_found_host) {
        printf("\nTempo total de busca: %.2f segundos\n", total_search_time);
        printf("Velocidade média: %s\n", formatSpeed(avg_speed));
        hipMemcpyAsync(private_key, streams[current_stream].d_private_key,
                      32, hipMemcpyDeviceToHost, streams[current_stream].stream);
        printf("Chave encontrada: ");
        for (int i = 0; i < 32; i++) {
            printf("%02x", private_key[i]);
        }
        printf("\n");
    } else {
        printf("\nChave não encontrada após %.2f segundos\n", total_search_time);
    }

    // Cleanup
    hipHostFree(host_private_key);
    hipHostFree(host_bitcoin_address);
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i].stream);
        hipFree(streams[i].d_private_key);
        hipFree(streams[i].d_bitcoin_address);
        hipFree(streams[i].d_match_found);
        hipStreamDestroy(streams[i].stream);
    }
    hipFree(d_target_address);
    freeGPUTables();

    return 0;
}